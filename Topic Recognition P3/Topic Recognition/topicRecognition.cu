#include "hip/hip_runtime.h"
// ConsoleApplication2.cpp : Defines the entry point for the console application.
//



#include<iostream>
#include<math.h>
#include<conio.h>
#include<stdlib.h>
#include<vector>
#include<algorithm>
#include<map>
#include<iterator>
#include <fstream>
#include <streambuf>
#include<string>
#include <dirent.h>
#include <boost/algorithm/string.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <random>


int D = 10000;
int N = 3;

std::vector<int> genRandomHV()
{
	std::vector<int> randomIndex(D);
	std::vector<int> randomHV(D);
	std::mt19937 r{ std::random_device{}() };
	if ((D % 2) == 1)
	{
		std::cout << "Dimension is odd";
	}
	else
	{
		for (int i = 0; i < D; i++)
			randomIndex[i] = i;
		std::shuffle(randomIndex.begin(), randomIndex.end(), r);
		for (int i = 0;i < D / 2;i++)
			randomHV[randomIndex[i]] = 1;
		for (int i = D / 2;i < D;i++)
			randomHV[randomIndex[i]] = -1;
	}
	
	return randomHV;
}

std::map<char, std::vector<int>> createItemMemory(std::map<char, std::vector<int>> iM)
{
	
		
	iM['a'] = genRandomHV();
	iM['b'] = genRandomHV();
	iM['c'] = genRandomHV();
	iM['d'] = genRandomHV();
	iM['e'] = genRandomHV();
	iM['f'] = genRandomHV();
	iM['g'] = genRandomHV();
	iM['h'] = genRandomHV();
	iM['i'] = genRandomHV();
	iM['j'] = genRandomHV();
	iM['k'] = genRandomHV();
	iM['l'] = genRandomHV();
	iM['m'] = genRandomHV();
	iM['n'] = genRandomHV();
	iM['o'] = genRandomHV();
	iM['p'] = genRandomHV();
	iM['q'] = genRandomHV();
	iM['r'] = genRandomHV();
	iM['s'] = genRandomHV();
	iM['t'] = genRandomHV();
	iM['u'] = genRandomHV();
	iM['v'] = genRandomHV();
	iM['w'] = genRandomHV();
	iM['x'] = genRandomHV();
	iM['y'] = genRandomHV();
	iM['z'] = genRandomHV();
	iM[char(32)] = genRandomHV();
	
	return iM;
}

std::vector<int> lookUpitemMemory(std::map<char, std::vector<int>> iM, char key)
{
	std::vector<int> randomHV(D);
	randomHV = iM[key];
	return randomHV;
}

double cosine_similarity(std::vector<int> A, std::vector<int> B)
{
	
	double dot = 0.0, denom_a = 0.0, denom_b = 0.0;
	for (int i = 0; i < D; ++i) {
		dot += A[i] * B[i];
		denom_a += A[i] * A[i];
		denom_b += B[i] * B[i];
	}
	return (dot / (sqrt(denom_a) * sqrt(denom_b)));
}

std::vector<int> binarizeHV(std::vector<int> langHV)
{
	int threshold = 0;
	/*for (size_t i = 0; i < langHV.size(); i++) {
		std::cout << langHV[i] << std::endl;
	}*/
	for (int i = 0; i < D; i++)
	{
		if (langHV[i] > threshold)
		{
			langHV[i] = 1;
		}
		else
		{
			langHV[i] = -1;
		}
	}
	
	/*for (size_t i = 0; i < langHV.size(); i++) {
		std::cout << langHV[i] << std::endl;
		}*/

	return langHV;
}

std::vector<int> computeSumHV(std::map<char, std::vector<int>> iM, size_t bufferSize, std::string  buffer)
{
	//std::vector<int> h_block0(D, 1);
	thrust::device_vector<int> st_block2(D, 1);
	thrust::device_vector<int> st_block3(D, 1);
	thrust::device_vector<int> st_block4(D, 1);
	thrust::device_vector<int> st_block5(D, 1);
	thrust::device_vector<int> st_block6(D, 1);
	thrust::device_vector<int> st_block7(D, 1);
	thrust::device_vector<int> block0(D, 1);
	thrust::device_vector<int> block1(D, 1);
	thrust::device_vector<int> block2(D, 1);
	thrust::device_vector<int> block3(D, 1);
	thrust::device_vector<int> block4(D, 1);
	thrust::device_vector<int> block5(D, 1);
	thrust::device_vector<int> block6(D, 1);
	thrust::device_vector<int> block7(D, 1);
	thrust::device_vector<int> nGrams(D, 1);
	thrust::device_vector <int > d_sumHV(D, 0);
	std::vector<int> sumHV(D, 0);
	if (N == 3)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];
			/*for (size_t i = 0; i < block0.size(); i++) {
			block0[i] = block1[i];
			}*/
			std::cout << key;

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			//rotate(h_block1.begin(), h_block1.end() - 1, h_block0.end());
			/*for (size_t i = 0; i < block1.size(); i++) {
			block1[i] = block2[i];
			}*/
			//block1 = block2; 
			//rotate(block1.begin(), block1.end() - 1, block1.end());
			st_block2 = lookUpitemMemory(iM, key);

			thrust::copy(st_block2.begin(), st_block2.end(), block2.begin());

			if (j >= 2)
			{
				thrust::copy(block2.begin(), block2.end(), nGrams.begin());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] = block2[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block1[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	std::cout << nGrams[i] << std::endl;
				//}
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block0[i];
				//}
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	sumHV[i] += nGrams[i];
				//}

			}

		}

	}

	else if (N == 4)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];
			/*for (size_t i = 0; i < block0.size(); i++) {
			block0[i] = block1[i];
			}*/
			//std::cout << key;

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			//rotate(h_block1.begin(), h_block1.end() - 1, h_block0.end());
			/*for (size_t i = 0; i < block1.size(); i++) {
			block1[i] = block2[i];
			}*/
			//block1 = block2; 
			//rotate(block1.begin(), block1.end() - 1, block1.end());
			st_block3 = lookUpitemMemory(iM, key);

			thrust::copy(st_block3.begin(), st_block3.end(), block3.begin());

			if (j >= 3)
			{
				thrust::copy(block3.begin(), block3.end(), nGrams.begin());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] = block2[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block1[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	std::cout << nGrams[i] << std::endl;
				//}
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block0[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());

				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	sumHV[i] += nGrams[i];
				//}

			}

		}

	}

	else if (N == 5)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];
			/*for (size_t i = 0; i < block0.size(); i++) {
			block0[i] = block1[i];
			}*/
			std::cout << key;

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			//rotate(h_block1.begin(), h_block1.end() - 1, h_block0.end());
			/*for (size_t i = 0; i < block1.size(); i++) {
			block1[i] = block2[i];
			}*/
			//block1 = block2; 
			//rotate(block1.begin(), block1.end() - 1, block1.end());
			st_block4 = lookUpitemMemory(iM, key);

			thrust::copy(st_block4.begin(), st_block4.end(), block4.begin());

			if (j >= 4)
			{
				thrust::copy(block4.begin(), block4.end(), nGrams.begin());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] = block2[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block1[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	std::cout << nGrams[i] << std::endl;
				//}
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block0[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());


				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	sumHV[i] += nGrams[i];
				//}

			}

		}

	}

	else if (N == 6)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];
			/*for (size_t i = 0; i < block0.size(); i++) {
			block0[i] = block1[i];
			}*/
			//std::cout << key;

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			thrust::copy(block5.begin() + 1, block5.end(), block4.begin());
			//rotate(h_block1.begin(), h_block1.end() - 1, h_block0.end());
			/*for (size_t i = 0; i < block1.size(); i++) {
			block1[i] = block2[i];
			}*/
			//block1 = block2; 
			//rotate(block1.begin(), block1.end() - 1, block1.end());
			st_block5 = lookUpitemMemory(iM, key);

			thrust::copy(st_block5.begin(), st_block5.end(), block5.begin());

			if (j >= 5)
			{
				thrust::copy(block5.begin(), block5.end(), nGrams.begin());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] = block2[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block4.begin(), thrust::multiplies<int>());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block1[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	std::cout << nGrams[i] << std::endl;
				//}
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block0[i];
				//}
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	sumHV[i] += nGrams[i];
				//}

			}

		}

	}

	else if (N == 7)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];
			/*for (size_t i = 0; i < block0.size(); i++) {
			block0[i] = block1[i];
			}*/
			//std::cout << key;

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			thrust::copy(block5.begin() + 1, block5.end(), block4.begin());
			thrust::copy(block6.begin() + 1, block6.end(), block5.begin());

			//rotate(h_block1.begin(), h_block1.end() - 1, h_block0.end());
			/*for (size_t i = 0; i < block1.size(); i++) {
			block1[i] = block2[i];
			}*/
			//block1 = block2; 
			//rotate(block1.begin(), block1.end() - 1, block1.end());
			st_block6 = lookUpitemMemory(iM, key);

			thrust::copy(st_block6.begin(), st_block6.end(), block6.begin());

			if (j >= 6)
			{
				thrust::copy(block6.begin(), block6.end(), nGrams.begin());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] = block2[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block5.begin(), thrust::multiplies<int>());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block1[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block4.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	std::cout << nGrams[i] << std::endl;
				//}
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block0[i];
				//}
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	sumHV[i] += nGrams[i];
				//}

			}

		}

	}

	else if (N == 8)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];
			/*for (size_t i = 0; i < block0.size(); i++) {
			block0[i] = block1[i];
			}*/
			//std::cout << key;

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			thrust::copy(block5.begin() + 1, block5.end(), block4.begin());
			thrust::copy(block6.begin() + 1, block6.end(), block5.begin());
			thrust::copy(block7.begin() + 1, block7.end(), block6.begin());

			//rotate(h_block1.begin(), h_block1.end() - 1, h_block0.end());
			/*for (size_t i = 0; i < block1.size(); i++) {
			block1[i] = block2[i];
			}*/
			//block1 = block2; 
			//rotate(block1.begin(), block1.end() - 1, block1.end());
			st_block7 = lookUpitemMemory(iM, key);

			thrust::copy(st_block7.begin(), st_block7.end(), block7.begin());

			if (j >= 7)
			{
				thrust::copy(block7.begin(), block7.end(), nGrams.begin());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] = block2[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block6.begin(), thrust::multiplies<int>());

				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block1[i];
				//}
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block5.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block4.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	std::cout << nGrams[i] << std::endl;
				//}
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	nGrams[i] *= block0[i];
				//}
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
				//for (size_t i = 0; i < nGrams.size(); i++) {
				//	sumHV[i] += nGrams[i];
				//}

			}

		}

	}
	thrust::copy(d_sumHV.begin(), d_sumHV.end(), sumHV.begin());

	return sumHV;

}

std::map<std::string, std::vector<int>> buildLanguage(std::map<char, std::vector<int>> iM)
{
	std::map< std::string, std::vector<int>> langAM;
	std::vector<int> langHV(D);
	//size_t size = 0;
	//char *ch = NULL;
	//FILE *file = NULL;
	/*errno_t err;*/

	std::string langLabels[8];
	langLabels[0] = "acq";
	langLabels[1] = "cru";
	langLabels[2] = "ear";
	langLabels[3] = "gra";
	langLabels[4] = "int";
	langLabels[5] = "mon";
	langLabels[6] = "shi";
	langLabels[7] = "tra";
	//std::string langText;
	//langText = "C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\acq.txt";
	/*langText[1] = "C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\cru.txt";
	langText[2] = "C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\gra.txt";
	langText[3] = "C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\int.txt";
	langText[4] = "C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\mon.txt";
	langText[5] = "C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\ear.txt";
	langText[6] = "C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\shi.txt";
	langText[7] = "C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\tra.txt";
	*/
	for (int i = 0; i < 8; i++)
	{

		/*int TempNumOne = langLabels[i].size();
		char Filename[100];
		for (int a = 0;a <= TempNumOne;a++)
		{
			Filename[a] = langText[a];
		}
		*/

		switch (i)
		{
		case 0: {std::ifstream t("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\acq.txt");
			std::string str;

			/*t.seekg(0, std::ios::end);
			str.reserve(t.tellg());
			t.seekg(0, std::ios::beg);*/

			/*std::ifstream t("file.txt");*/
			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);

			/*fseek(file, 0, SEEK_END);
			size = ftell(file);
			rewind(file);
			ch = (char *)malloc((size + 1) * sizeof(ch));
			fread(ch, size, 1, file);*/
			std::cout << "Training File:" << langLabels[i] << std::endl;

			langHV = computeSumHV(iM, size, buffer);
			langAM[(langLabels[i])] = binarizeHV (langHV);
			//langAM[(langLabels[i])] = binarizeHV(langAM[(langLabels[i])]);
			break;
		}
		case 1: {
			std::ifstream t("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\cru.txt");
			std::string str;

			/*t.seekg(0, std::ios::end);
			str.reserve(t.tellg());
			t.seekg(0, std::ios::beg);*/

			/*std::ifstream t("file.txt");*/
			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);

			/*fseek(file, 0, SEEK_END);
			size = ftell(file);
			rewind(file);
			ch = (char *)malloc((size + 1) * sizeof(ch));
			fread(ch, size, 1, file);*/
			std::cout << "Training File:" << langLabels[i] << std::endl;

			langHV = computeSumHV(iM, size, buffer);
			langAM[(langLabels[i])] = binarizeHV(langHV);
			//langAM[(langLabels[i])] = binarizeHV(langAM[(langLabels[i])]);
			break;
		}
		case 2: {
			std::ifstream t("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\ear.txt");
			std::string str;

			/*t.seekg(0, std::ios::end);
			str.reserve(t.tellg());
			t.seekg(0, std::ios::beg);*/

			/*std::ifstream t("file.txt");*/
			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);

			/*fseek(file, 0, SEEK_END);
			size = ftell(file);
			rewind(file);
			ch = (char *)malloc((size + 1) * sizeof(ch));
			fread(ch, size, 1, file);*/
			std::cout << "Training File:" << langLabels[i] << std::endl;

			langHV = computeSumHV(iM, size, buffer);
			langAM[(langLabels[i])] = binarizeHV(langHV);
			//langAM[(langLabels[i])] = binarizeHV(langAM[(langLabels[i])]);
			break;
		}
		case 3: {
			std::ifstream t("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\gra.txt");
			std::string str;

			/*t.seekg(0, std::ios::end);
			str.reserve(t.tellg());
			t.seekg(0, std::ios::beg);*/

			/*std::ifstream t("file.txt");*/
			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);

			/*fseek(file, 0, SEEK_END);
			size = ftell(file);
			rewind(file);
			ch = (char *)malloc((size + 1) * sizeof(ch));
			fread(ch, size, 1, file);*/
			std::cout << "Training File:" << langLabels[i] << std::endl;

			langHV = computeSumHV(iM, size, buffer);
			langAM[(langLabels[i])] = binarizeHV(langHV);
			//langAM[(langLabels[i])] = binarizeHV(langAM[(langLabels[i])]);
			break;
		}
		case 4: {
			std::ifstream t("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\int.txt");
			std::string str;

			/*t.seekg(0, std::ios::end);
			str.reserve(t.tellg());
			t.seekg(0, std::ios::beg);*/

			/*std::ifstream t("file.txt");*/
			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);

			/*fseek(file, 0, SEEK_END);
			size = ftell(file);
			rewind(file);
			ch = (char *)malloc((size + 1) * sizeof(ch));
			fread(ch, size, 1, file);*/
			std::cout << "Training File:" << langLabels[i] << std::endl;

			langHV = computeSumHV(iM, size, buffer);
			langAM[(langLabels[i])] = binarizeHV(langHV);
			//langAM[(langLabels[i])] = binarizeHV(langAM[(langLabels[i])]);
			break;
		}
		case 5: {
			std::ifstream t("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\mon.txt");
			std::string str;

			/*t.seekg(0, std::ios::end);
			str.reserve(t.tellg());
			t.seekg(0, std::ios::beg);*/

			/*std::ifstream t("file.txt");*/
			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);

			/*fseek(file, 0, SEEK_END);
			size = ftell(file);
			rewind(file);
			ch = (char *)malloc((size + 1) * sizeof(ch));
			fread(ch, size, 1, file);*/
			std::cout << "Training File:" << langLabels[i] << std::endl;

			langHV = computeSumHV(iM, size, buffer);
			langAM[(langLabels[i])] = binarizeHV(langHV);
			//langAM[(langLabels[i])] = binarizeHV(langAM[(langLabels[i])]);
			break;
		}
		case 6: {
			std::ifstream t("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\shi.txt");
			std::string str;

			/*t.seekg(0, std::ios::end);
			str.reserve(t.tellg());
			t.seekg(0, std::ios::beg);*/

			/*std::ifstream t("file.txt");*/
			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);

			/*fseek(file, 0, SEEK_END);
			size = ftell(file);
			rewind(file);
			ch = (char *)malloc((size + 1) * sizeof(ch));
			fread(ch, size, 1, file);*/
			std::cout << "Training File:" << langLabels[i] << std::endl;

			langHV = computeSumHV(iM, size, buffer);
			langAM[(langLabels[i])] = binarizeHV(langHV);
			//langAM[(langLabels[i])] = binarizeHV(langAM[(langLabels[i])]);
			break;
		}
		case 7: {
			std::ifstream t("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\tra.txt");
			std::string str;

			/*t.seekg(0, std::ios::end);
			str.reserve(t.tellg());
			t.seekg(0, std::ios::beg);*/

			/*std::ifstream t("file.txt");*/
			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);

			/*fseek(file, 0, SEEK_END);
			size = ftell(file);
			rewind(file);
			ch = (char *)malloc((size + 1) * sizeof(ch));
			fread(ch, size, 1, file);*/
			std::cout << "Training File:" << langLabels[i] << std::endl;

			langHV = computeSumHV(iM, size, buffer);
			langAM[(langLabels[i])] = binarizeHV(langHV);
			//langAM[(langLabels[i])] = binarizeHV(langAM[(langLabels[i])]);
			break;
		}

				/*}
					std::ifstream t("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\acq.txt");
					std::string str;

					t.seekg(0, std::ios::end);
					str.reserve(t.tellg());
					t.seekg(0, std::ios::beg);

					std::ifstream t("file.txt");
					t.seekg(0, std::ios::end);
					size_t size = t.tellg();
					std::string buffer(size, ' ');
					t.seekg(0);
					t.read(&buffer[0], size);

					/*fseek(file, 0, SEEK_END);
					size = ftell(file);
					rewind(file);
					ch = (char *)malloc((size + 1) * sizeof(ch));
					fread(ch, size, 1, file);
					std::cout << size;

					langHV = computeSumHV(iM, size, buffer);
					langAM[(langLabels[i])] = langHV;

					/*printf("%s\n", buffer.c_str());*/

		}
	}
	return langAM;

	/*
	for(int i=0;i<size;i++)
	{
	printf("%c",ch[i]);
	}
	*/
	
	
}


double test(std::map<char, std::vector<int>> iM, std::map<std::string, std::vector<int>> langAM)
{
	double total = 0.0;
	double correct = 0.0;
	double accuracy = 0;
	double maxAngle, angle = 0;
	std::string predictLang;
	std::vector<int> textHV;

	std::string langLabels[8];
	langLabels[0] = "acq";
	langLabels[1] = "cru";
	langLabels[2] = "gra";
	langLabels[3] = "int";
	langLabels[4] = "mon";
	langLabels[5] = "ear";
	langLabels[6] = "shi";
	langLabels[7] = "tra";

	DIR *pdir = NULL; // remember, it's good practice to initialise a pointer to NULL!

	    pdir = opendir ("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\ConsoleApplication2\\ConsoleApplication2\\Debug\\testing_texts"); // "." will refer to the current directory

	    struct dirent *pent = NULL;

	 

	    // I used the current directory, since this is one which will apply to anyone reading

	    // this tutorial~ If I said "C:\\" and you're on Linux, it may get a little confusing!

	    if (pdir == NULL) // if pdir wasn't initialised correctly

	    { // print an error message and exit the program

	        std::cout << "\nERROR! pdir could not be initialised correctly";

	        exit (3);

	    } // end if

	 

	    while (pent = readdir (pdir)) // while there is still something in the directory to list

	    {

	        if (pent == NULL) // if pent has not been initialised correctly

	        { // print an error message, and exit the program

	            std::cout << "\nERROR! pent could not be initialised correctly";

	            exit (3);

	        }
			if (strcmp(pent->d_name, ".") != 0 && strcmp(pent->d_name, "..") != 0)
	        // otherwise, it was initialised correctly. Let's print it on the console:
			//if (pent->d_name == ".")
			//{
				//continue;
			//}
			//else
			{
				std::cout << pent->d_name << std::endl;
				std::string name = pent->d_name;
				std::string actualLabel = name.substr(0, 3);
				std::vector<std::string> list{ "C:", "Users", "Mohammed Aashyk", "Documents", "Visual Studio 2015", "Projects", "ConsoleApplication2", "ConsoleApplication2", "Debug", "testing_texts", name };
				//std::cout << name;
				std::string joined = boost::algorithm::join(list, "\\");
				//std::cout << joined;
				std::ifstream t(joined);
				std::string str;

				/*t.seekg(0, std::ios::end);
				str.reserve(t.tellg());
				t.seekg(0, std::ios::beg);*/

				/*std::ifstream t("file.txt");*/
				t.seekg(0, std::ios::end);
				size_t size = t.tellg();
				std::string buffer(size, ' ');
				t.seekg(0);
				t.read(&buffer[0], size);

				/*fseek(file, 0, SEEK_END);
				size = ftell(file);
				rewind(file);
				ch = (char *)malloc((size + 1) * sizeof(ch));
				fread(ch, size, 1, file);*/
				

				std::cout << "Loading test file:" << pent->d_name << std::endl;
				textHV = computeSumHV(iM, size, buffer);
				textHV = binarizeHV(textHV);
				maxAngle = -1;
				for (int i = 0; i < 8; i++)
				{
					angle = cosine_similarity(langAM[langLabels[i]], textHV);
					if (angle > maxAngle)
					{
						maxAngle = angle;
						predictLang = langLabels[i];
					}
					

				}
				if (predictLang == actualLabel)
				{
					correct = correct + 1.0;
				}
				else
				{
					std::cout << predictLang << "  -->  " << actualLabel <<  std::endl;
				}
			}

			total = total + 1.0;
	    }

		//std::cout << pent->d_name;
		
		

	    // finally, let's close the directory

	    closedir (pdir);

	 

	    //std::cin.get (); // pause for input

	   // return EXIT_SUCCESS; // everything went OK
		accuracy = correct / total * 100;

		return accuracy;
}


void printPair(const std::pair<char, std::vector<int> > &p)
{
	std::cout << "Key: " << p.first << std::endl;
	copy(p.second.begin(), p.second.end(), std::ostream_iterator<int>(std::cout, "\t"));
}
int main()
{
	
	std::vector<int> rand;
	std::map<char, std::vector<int>> iM;
	std::map<std::string, std::vector<int>> langAM;
	double correct;
	iM = createItemMemory(iM);

	//for_each(iM.begin(), iM.end(), printPair);
	//std::cout << "rand contains:";
	//for (std::vector<int>::iterator it = rand.begin(); it != rand.end(); ++it)
	//	std::cout << ' ' << *it;

	langAM = buildLanguage(iM);
	/*std::map<std::string, std::vector<int>>::iterator pos;
	for (pos = langAM.begin(); pos != langAM.end(); ++pos) {
	std::cout << "key: \"" << pos->first << "\" " << std::endl << "values: \"" ;
	typedef std::vector<int>::const_iterator ListIterator;
	for (ListIterator list_iter = pos->second.begin(); list_iter != pos->second.end(); list_iter++)
	std::cout << " " << *list_iter << std::endl;
	}
	/* for (std::map<std::string, std::vector<long int>> ::const_iterator it = langAM.begin();//
	it != langAM.end(); ++it)
	{
	std::cout << it->first << " " << it->second.first << " " << it->second.second << "\n";
	}*/
	
	correct = test(iM, langAM);

	/*std::map<std::string, std::vector<int>>::iterator pos;
	for (pos = langAM.begin(); pos != langAM.end(); ++pos) {
		std::cout << "key: \"" << pos->first << "\" " << std::endl << "values: \"" ;
		typedef std::vector<int>::const_iterator ListIterator;
		for (ListIterator list_iter = pos->second.begin(); list_iter != pos->second.end(); list_iter++)
			std::cout << " " << *list_iter << std::endl;
	}
	/* for (std::map<std::string, std::vector<long int>> ::const_iterator it = langAM.begin();
		it != langAM.end(); ++it)
	{
		std::cout << it->first << " " << it->second.first << " " << it->second.second << "\n";
	}*/
	
	std::cout << correct << "%" << std::endl << "Run Success!";


}



